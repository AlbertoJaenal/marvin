// Please choose a data type to compile
#define DATATYPE 0
#include "../../marvin.hpp"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <fstream>

int main(int argc, char **argv){
    if (argc != 5){
        std::cout<<"Usage:"<<std::endl;
        std::cout<<argv[0]<<" network.json model1.marvin[,model2.marvin,...] categories.txt webcamID"<<std::endl;
        return 0;
    }

    // initialize the network
    marvin::Net net(argv[1]);
    net.Malloc(marvin::Testing);
    std::vector<std::string> models = marvin::getStringVector(argv[2]);
    for (int m=0;m<models.size();++m)   net.loadWeights(models[m]);
    // where the data is
    marvin::Response* rData = net.responses[0];
    marvin::Response* rResult = net.responses[net.responses.size()-1];
    marvin::Tensor<StorageT>* cpuResult = new marvin::Tensor<StorageT>(rResult->dim);

    // image net list
    std::vector<std::string> objectCategories;
    objectCategories.resize(1000);
    std::ifstream fclass(argv[3]);
    for(int i=0;i<1000;++i){
        std::getline(fclass,objectCategories[i]);
    }
    fclass.close();


    // initialize video data feed from webcam

    cv::VideoCapture capture; //try to open string, this will attempt to open it as a video file or image sequence
    capture.open(atoi(argv[4]));

    int width_webcome = 640;
    int height_webcome = 480;

    capture.set(CV_CAP_PROP_FRAME_WIDTH,width_webcome);
    capture.set(CV_CAP_PROP_FRAME_HEIGHT,height_webcome);    
    if (!capture.isOpened()) {
        std::cerr << "Failed to open the video device, video file or image sequence!\n" << std::endl;
        return 1;
    }
    cv::Mat image_original;
    cv::Mat image_resize;

    marvin::PlaceHolderDataLayer* pDataLayer = (marvin::PlaceHolderDataLayer*)net.layers[0];

    int height_network = pDataLayer->dim[2];
    int width_network = pDataLayer->dim[3];
    int numel_network = width_network*height_network*3;

    uint8_t* imageGPU_OCV;  marvin::checkCUDA(__LINE__, hipMalloc(&imageGPU_OCV, width_network*height_network*3));
    uint8_t* imageGPU;      marvin::checkCUDA(__LINE__, hipMalloc(&imageGPU, width_network*height_network*3));

    std::cout<<"====================================================================================================================================="<<std::endl;

    // while it is running
    while (true){
        // read image data, e.g. using OpenCV to get an image from webcam
        capture >> image_original;
        if (image_original.empty()) break;

        // resize image for the network
        cv::resize(image_original, image_resize, cv::Size(height_network,width_network));

        // copy the image from CPU to GPU
        hipMemcpy(imageGPU_OCV, image_resize.data, height_network*width_network*3*sizeof(uint8_t), hipMemcpyHostToDevice);

        // convert the color image from OpenCV format (BGR with channel first) to Marvin format (CHW with RGB)
        marvin::OpenCV_BGR_image_to_Marvin(3, width_network, width_network, imageGPU_OCV, imageGPU);

        // convert image from uint8_t to StorageT on GPU
        marvin::Kernel_convert_to_StorageT_subtract<<<marvin::CUDA_GET_BLOCKS(numel_network), CUDA_NUM_THREADS >>>(marvin::CUDA_GET_LOOPS(numel_network), numel_network, numel_network, imageGPU, pDataLayer->meanGPU, rData->dataGPU);

        // test the network
        net.forward();

        // read the result from GPU to CPU
        cpuResult->readGPU(rResult->dataGPU);

        // visualize the result or use the result
        imshow("Marvin webcam demo", image_resize);
        int iMax = 0;
        ComputeT vMax = 0;
        for(int i=0;i<1000;i++){
            ComputeT v = CPUStorage2ComputeT(cpuResult->CPUmem[i]);
            if (v>vMax){
                vMax = v;
                iMax = i;
            }
        }
        std::cout<<objectCategories[iMax]<<std::endl;
        //std::vector<int> display_dim = {10};
        //cpuResult->print(display_dim);

        // any keyboard input
        char key = (char)cv::waitKey(30);
        if (key=='q' || key=='Q' || key==27) break;
    }

    marvin::checkCUDA(__LINE__, hipFree(imageGPU_OCV));
    marvin::checkCUDA(__LINE__, hipFree(imageGPU));

    delete cpuResult;

    return 0;
}

