// Please choose a data type to compile
#define DATATYPE 0
#include "../../marvin.hpp"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <fstream>

int main(int argc, char **argv){
    if (argc != 5){
        std::cout<<"Usage:"<<std::endl;
        std::cout<<argv[0]<<" network.json model1.marvin[,model2.marvin,...] categories.txt webcamID"<<std::endl;
        return 0;
    }

    // initialize the network
    marvin::Net net(argv[1]);
    net.Malloc(marvin::Testing);
    std::vector<std::string> models = marvin::getStringVector(argv[2]);
    for (int m=0;m<models.size();++m)   net.loadWeights(models[m]);
    // where the data is
    marvin::Response* rData = net.responses[0];
    marvin::Response* rResult = net.responses[net.responses.size()-1];

    // image net list
    std::vector<std::string> objectCategories;
    objectCategories.resize(1000);
    std::ifstream fclass(argv[3]);
    for(int i=0;i<1000;++i){
        std::getline(fclass,objectCategories[i]);
    }
    fclass.close();


    // initialize video data feed from webcam

    cv::VideoCapture capture; //try to open string, this will attempt to open it as a video file or image sequence
    capture.open(atoi(argv[4]));

    int width_webcam = 640;
    int height_webcam = 480;

    capture.set(CV_CAP_PROP_FRAME_WIDTH,width_webcam);
    capture.set(CV_CAP_PROP_FRAME_HEIGHT,height_webcam);    
    if (!capture.isOpened()) {
        std::cerr << "Failed to open the video device, video file or image sequence!\n" << std::endl;
        return 1;
    }


    marvin::PlaceHolderDataLayer* pDataLayer = (marvin::PlaceHolderDataLayer*)net.layers[0];

    int height_network = pDataLayer->dim[2];
    int width_network = pDataLayer->dim[3];
    int numel_network = width_network*height_network*3;

    //uint8_t* imageGPU_OCV;  marvin::checkCUDA(__LINE__, hipMalloc(&imageGPU_OCV, width_network*height_network*3));

    uint8_t* image_resize_memCPU;
	marvin::checkCUDA(__LINE__, hipHostAlloc( (void**)&image_resize_memCPU, width_network*height_network*3*sizeof(uint8_t), hipHostMallocWriteCombined | hipHostMallocMapped ));

	uint8_t* image_resize_memGPU;	hipHostGetDevicePointer( &image_resize_memGPU, image_resize_memCPU, 0 );

    cv::Mat image_original;
    cv::Mat image_resize(height_network,width_network,CV_8UC3,image_resize_memCPU);

    uint8_t* imageGPU;      marvin::checkCUDA(__LINE__, hipMalloc(&imageGPU, width_network*height_network*3));


    // allocate CPU for the host
    StorageT* rResult_CPU; 
	marvin::checkCUDA(__LINE__, hipHostAlloc( (void**)&rResult_CPU, rResult->numBytes(), hipHostMallocWriteCombined | hipHostMallocMapped ));
    marvin::Tensor<StorageT>* cpuResult = new marvin::Tensor<StorageT>(rResult->dim, rResult_CPU);

    // replace the original GPU memory
    marvin::checkCUDA(__LINE__, hipFree(rResult->dataGPU));
	hipHostGetDevicePointer( &(rResult->dataGPU), rResult_CPU, 0 );

    std::cout<<"====================================================================================================================================="<<std::endl;

    // while it is running
    while (true){
        // read image data, e.g. using OpenCV to get an image from webcam
        marvin::tic();
        capture >> image_original;
        std::cout<<"capture image: ";
        marvin::toc();
        if (image_original.empty()) break;

        // resize image for the network
        marvin::tic();
        cv::resize(image_original, image_resize, cv::Size(height_network,width_network));
        std::cout<<"resize image: ";
        marvin::toc();

        // copy the image from CPU to GPU
        //hipMemcpy(imageGPU_OCV, image_resize.data, height_network*width_network*3*sizeof(uint8_t), hipMemcpyHostToDevice);

        // convert the color image from OpenCV format (BGR with channel first) to Marvin format (CHW with RGB)
        marvin::tic();
        marvin::OpenCV_BGR_image_to_Marvin(3, height_network, width_network, image_resize_memGPU, imageGPU);
        std::cout<<"OpenCV_BGR_image_to_Marvin: ";
        marvin::toc();

        // convert image from uint8_t to StorageT on GPU
        marvin::tic();
        marvin::Kernel_convert_to_StorageT_subtract<<<marvin::CUDA_GET_BLOCKS(numel_network), CUDA_NUM_THREADS >>>(marvin::CUDA_GET_LOOPS(numel_network), numel_network, numel_network, imageGPU, pDataLayer->meanGPU, rData->dataGPU);
        std::cout<<"Kernel_convert_to_StorageT_subtract: ";
        marvin::toc();

        // test the network
        marvin::tic();
        net.forward();
        std::cout<<"net.forward(): ";
        marvin::toc();

        // read the result from GPU to CPU
        //marvin::tic();
        //cpuResult->readGPU(rResult->dataGPU);
        //std::cout<<"cpuResult->readGPU: ";
        //marvin::toc();

        // visualize the result or use the result
        imshow("Marvin webcam demo", image_resize);
        int iMax = 0;
        ComputeT vMax = 0;
        for(int i=0;i<1000;i++){
            ComputeT v = CPUStorage2ComputeT(cpuResult->CPUmem[i]);
            if (v>vMax){
                vMax = v;
                iMax = i;
            }
        }
        std::cout<<objectCategories[iMax]<<std::endl;
        //std::vector<int> display_dim = {10};
        //cpuResult->print(display_dim);

        // any keyboard input
        char key = (char)cv::waitKey(30);
        if (key=='q' || key=='Q' || key==27) break;
    }

    marvin::checkCUDA(__LINE__, hipFree(image_resize_memGPU));
    free(image_resize_memCPU);
    marvin::checkCUDA(__LINE__, hipFree(imageGPU));

    delete cpuResult;

    return 0;
}

